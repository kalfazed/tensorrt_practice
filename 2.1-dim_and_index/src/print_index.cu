#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void print_idx_kernel(){
  printf("block idx: (%3d, %3d, %3d), thread idx: (%3d, %3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ void print_dim_kernel(){
  printf("grid dimension: (%3d, %3d, %3d), thread dimension: (%3d, %3d, %3d)\n",
         gridDim.z, gridDim.y, gridDim.x,
         blockDim.z, blockDim.y, blockDim.x);
}

__global__ void print_thread_idx_per_block_kernel(){
  int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

  printf("block idx: (%3d, %3d, %3d), thread idx: %3d\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index);
}

__global__ void print_thread_idx_kernel(){
  int bSize  = blockDim.z * blockDim.y * blockDim.x;

  int bIndex = blockIdx.z * gridDim.x * gridDim.y + \
               blockIdx.y * gridDim.x + \
               blockIdx.x;

  int tIndex = threadIdx.z * blockDim.x * blockDim.y + \
               threadIdx.y * blockDim.x + \
               threadIdx.x;

  int index  = bIndex * bSize + tIndex;

  printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n", 
         bIndex, tIndex, index);
}

void print_one_dim(){
  int inputSize = 32;

  int blockDim = 4;
  int gridDim = inputSize / blockDim;

  dim3 block(blockDim);
  dim3 grid(gridDim);

  print_idx_kernel<<<grid, block>>>();
  print_dim_kernel<<<grid, block>>>();
  print_thread_idx_per_block_kernel<<<grid, block>>>();
  print_thread_idx_kernel<<<grid, block>>>();

  hipDeviceSynchronize();
}

void print_two_dim(){
  int inputWidth = 8;

  int blockDim = 2;
  int gridDim = inputWidth / blockDim;

  dim3 block(blockDim, blockDim);
  dim3 grid(gridDim, gridDim);

  print_idx_kernel<<<grid, block>>>();
  // print_dim_kernel<<<grid, block>>>();
  // print_thread_idx_per_block_kernel<<<grid, block>>>();
  // print_thread_idx_kernel<<<grid, block>>>();

  hipDeviceSynchronize();
}

int main() {
  /*
    synchronize是同步的意思，有几种synchronize

    cudaDeviceSynchronize: cpu端停止执行，知道gpu端完成这个语句以前的所有cuda操作
    cudaStreamSynchronize: 跟cudaDeviceSynchronize很像，但是这个是针对某一个stream的。只同步指定的stream中的cpu/gpu操作，其他的不管
    cudaThreadSynchronize: 现在已经不被推荐使用的方法
    __syncthreads:         线程块内同步
  */
  // print_one_dim();
  print_two_dim();
  return 0;
}
